
#include <hip/hip_runtime.h>
#include <float.h>
#include <iostream>
#include <fstream>
#include <math.h>
#include <cmath>
#include <algorithm>
#include <vector>
#include <dirent.h>
#include <chrono>
#include <string>
// #include <opencv2/opencv.hpp>


__global__ void relu(float* input, float* output, int N, int M){
    int i = threadIdx.y;
    int j = threadIdx.x;
    output[i*N + j] = fmaxf(0.0, input[i*N + j]);
}


__global__ void tanh(float* input, float* output, int N, int M){
    int i = threadIdx.y;
    int j = threadIdx.x;
    output[i*N + j] = tanh(input[i*N + j]);
}



__global__ void maxPooling(float* input, float* output, int N, int M, int output_size){
    int i = threadIdx.y;
    int j = threadIdx.x;
    float max = FLT_MIN;
    for (int k = 0; k < M; k++){
        for (int l = 0; l < M; l++){
            int ind = (i+k)*N + j+l;
            max = fmaxf(max, input[ind]);
        }
    }
    output[i*output_size + j] = max;
}


__global__ void avgPooling(float* input, float* output, int N, int M, int output_size){
    int i = threadIdx.y;
    int j = threadIdx.x;
    float sum = 0.0;
    for (int k = 0; k < M; k++){
        for (int l = 0; l < M; l++){
            sum += input[(i+k)*N + j+l];
        }
    }
    output[i*output_size + j] = (float)(sum/(M*M));
}

__global__ void convolution(float* input, float* weights, float* output, int N, int M, int P){
    int output_size = N+2*P-M+1;
    // only one block
    int i = threadIdx.y;
    int j = threadIdx.x;
    float sum = 0.0f;
    for(int k = 0; k < M; k++){
        for (int l = 0; l < M; l++){
            sum += input[(i+k)*(N+2*P) + j+l] * weights[k*M + l];
        }
    }
    output[i*output_size + j] = sum;

}




__global__ void sigmoid(float* input, float* output, int N){
    int i = threadIdx.x;
    output[i] = 1/(1+exp(-input[i]));
}

void softmax(float* input, float* output, int N){
    float sum = 0.0;
    for (int i = 0; i < N; i++){
        output[i] = exp(input[i]);
        sum += output[i];
    }
    for (int i = 0; i < N; i++){
        output[i] = output[i]/sum;
    }
}


int main(int argc, char *argv[]){
    if (std::stoi(argv[1]) == 1){ 
        //convolution

        int N = std::stoi(argv[2]);
        int M = std::stoi(argv[3]);
        int P = std::stoi(argv[4]);
        // N, M matrices are float matrices
        float* input = new float[(N+2*P)*(N+2*P)];
        float* weights = new float[M*M];
        float* output = new float[(N+2*P-M+1)*(N+2*P-M+1)];


        int argvIndex = 5;

        for (int i = 0; i < (N+2*P)*(N+2*P); i++){
            if (i < P*(N+2*P)){
                input[i] = 0.0;
            }
            else if (i >= (N+P)*(N+2*P)){
                input[i] = 0.0;
            }
            else if (i % (N+2*P) < P){
                input[i] = 0.0;
            }
            else if (i % (N+2*P) >= N+P){
                input[i] = 0.0;
            }
            else{
                input[i] = std::stof(argv[argvIndex]);
                argvIndex++;
            }
        }

        for(int i = 0; i < M*M; i++){
            weights[i] = std::stof(argv[argvIndex]);
            argvIndex++;
        }

        float* input_d;
        float* weights_d;
        float* output_d;
        hipMalloc(&input_d, (N+2*P)*(N+2*P)*sizeof(float));
        hipMalloc(&weights_d, M*M*sizeof(float));
        hipMalloc(&output_d, (N+2*P-M+1)*(N+2*P-M+1)*sizeof(float));
        hipMemcpy(input_d, input, (N+2*P)*(N+2*P)*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(weights_d, weights, M*M*sizeof(float), hipMemcpyHostToDevice);
        convolution<<<1, dim3(N+2*P-M+1, N+2*P-M+1)>>>(input_d, weights_d, output_d, N, M, P);

        hipMemcpy(output, output_d, (N+2*P-M+1)*(N+2*P-M+1)*sizeof(float), hipMemcpyDeviceToHost);

        //print as matrix
        for (int i = 0; i < N+2*P-M+1; i++){
            for (int j = 0; j < N+2*P-M+1; j++){
                std::cout << output[i*(N+2*P-M+1) + j] << " ";
            }
            std::cout << std::endl;
        }
        hipFree(input_d);
        hipFree(weights_d);
        hipFree(output_d);
        delete[] input;
        delete[] output;
        delete[] weights;
    }

    if(std::stoi(argv[1]) == 2){
        //activation 0 = relu, 1 = tanh
        int activation = std::stoi(argv[2]);
        int N = std::stoi(argv[3]);
        int M = std::stoi(argv[4]);
        float* input = new float[N*M];
        float* output = new float[N*M];
        for (int i = 0; i < N*M; i++){
            input[i] = std::stof(argv[i+5]);
        }

        float* input_d;
        float* output_d;
        hipMalloc(&input_d, N*M*sizeof(float));
        hipMalloc(&output_d, N*M*sizeof(float));

        hipMemcpy(input_d, input, N*M*sizeof(float), hipMemcpyHostToDevice);


        if (activation == 0){
            relu<<<1, dim3(N,M)>>>(input_d, output_d, N, M);
        }
        else if (activation == 1){
            tanh<<<1, dim3(N,M)>>>(input_d, output_d, N, M);
        }

        hipMemcpy(output, output_d, N*M*sizeof(float), hipMemcpyDeviceToHost);
        //print as matrix
        for (int i = 0; i < N; i++){
            for (int j = 0; j < M; j++){
                std::cout << output[i*M + j] << " ";
            }
            std::cout << std::endl;
        }
        hipFree(input_d);
        hipFree(output_d);

        delete[] input;
        delete[] output;
    }

    if(std::stoi(argv[1]) == 3){
        int pool_func = std::stoi(argv[2]);
        int M = std::stoi(argv[3]);
        int N = std::stoi(argv[4]);
        float* input = new float[N*N];
        for(int i = 0; i < N*N; i++){
            input[i] = std::stof(argv[i+5]);
        }
        // ceil of N/M
        int output_size = N-M+1;
        float* output = new float[output_size*output_size];

        float* input_d;
        float* output_d;
        hipMalloc(&input_d, N*N*sizeof(float));
        hipMalloc(&output_d, output_size*output_size*sizeof(float));
        hipMemcpy(input_d, input, N*N*sizeof(float), hipMemcpyHostToDevice);
        if (pool_func == 0){
            maxPooling<<<1, dim3(output_size, output_size)>>>(input_d, output_d, N, M, output_size);
        }
        else if (pool_func == 1){
            avgPooling<<<1, dim3(output_size, output_size)>>>(input_d, output_d, N, M, output_size);
        }
        hipMemcpy(output, output_d, output_size*output_size*sizeof(float), hipMemcpyDeviceToHost);
        //print the output matrix
        for (int i = 0; i < output_size; i++){
            for (int j = 0; j < output_size; j++){
                std::cout << output[i*output_size + j] << " ";
            }
            std::cout << std::endl;
        }
        hipFree(input_d);
        hipFree(output_d);

        delete[] input;
        delete[] output;
        
    }
    if(std::stoi(argv[1]) == 4){
        float* input = new float[argc - 3];
        int function = std::stoi(argv[2]);
        for (int i = 0; i < argc - 3; i++){
            input[i] = std::stof(argv[i+3]);
        }

        int N = argc - 3;
        float* output = new float[N];
        // 0 = sigmoid, 1 = softmax
        if (function == 0){
            float* input_d;
            float* output_d;
            hipMalloc(&input_d, N*sizeof(float));
            hipMalloc(&output_d, N*sizeof(float));
            hipMemcpy(input_d, input, N*sizeof(float), hipMemcpyHostToDevice);
            sigmoid<<<1, N>>>(input_d, output_d, N);
            hipMemcpy(output, output_d, N*sizeof(float), hipMemcpyDeviceToHost);
            hipFree(input_d);
            hipFree(output_d);

        }
        else if (function == 1){
            softmax(input,output, N);
        }
        for (int i = 0; i < N; i++){
            std::cout << output[i] << " ";
        }
        std::cout << std::endl;
        delete[] input;
        delete[] output;
    }
}